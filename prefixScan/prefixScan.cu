#include "hip/hip_runtime.h"
#include "prefixScan.hpp"

__global__
void prefixScanSimpleGPUImpl(const float * indata, float * outdata, size_t size) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if(index == 0) {
        outdata[index] = 0;
        return;
    }

    for(int i = 0 ; i < index ; i++) {
        outdata[index] += indata[i];
    }
}

void prefixScanSimpleGPU(const float * indata, float * outdata, size_t size) {
    prefixScanSimpleGPUImpl<<<1024, 1024>>>(indata, outdata, size);
}
