#include "hip/hip_runtime.h"
#include "prefixScan.hpp"

__global__
void prefixScanSimpleGPUImpl(const float * indata, float * outdata, std::size_t size) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int iStride = index; iStride < size ; iStride += stride) {
        outdata[iStride] = 0;

        for(int i = 0 ; i < iStride ; i++) {
            outdata[iStride] += indata[i];
        }
    }
}

void prefixScanSimpleGPU(const float * indata, float * outdata, std::size_t size) {
    const std::size_t bytes = size * sizeof(float);
    float * d_indata;
    float * d_outdata;

    hipMalloc(&d_indata, bytes);
    hipMalloc(&d_outdata, bytes);
    hipMemcpy(d_indata, indata, bytes, hipMemcpyHostToDevice);

    prefixScanSimpleGPUImpl<<<1, 1024>>>(d_indata, d_outdata, size);
    hipMemcpy(outdata, d_outdata, bytes, hipMemcpyDeviceToHost);

    hipFree(d_indata);
    hipFree(d_outdata);
}
