
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void square(float *arr, int size) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if(index < size) {
        arr[index] = (float)(index * index);
    }
}

int main() {
    constexpr int size = 10;
    float arr[size] = {0};
    float *d_arr;

    hipMalloc(&d_arr, size * sizeof(float));
    square<<<1, size>>>(d_arr, size);
    hipMemcpy(arr, d_arr, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_arr);

    for(int i = 0 ; i < size ; i++) {
        std::cout << arr[i] << ";";
    }
    std::cout << std::endl;

    return 0;
}